#include "hip/hip_runtime.h"
﻿#include "Tensor.cuh"
#include <hipDNN.h>


template <typename T>
Tensor<T>::Tensor(int n, int c, int h, int w) :N(n), C(c), H(h), W(w)
{

	Strides[0] = 1;
	Strides[1] = W;
	Strides[2] = H * W;
	Strides[3] = C * H * W;

	IsChunkPart = false;
	IsOwnData = true;

	TotalSize = sizeof(T) * N * C * H * W;

	hipMalloc(&Data, TotalSize);

	std::vector<T*> hostPtrs(N);
	for (size_t i = 0; i < N; ++i)
		hostPtrs[i] = Data + i * Strides[3];

	hipMalloc(&BatchPtrs, N * sizeof(T*));
	hipMemcpy(BatchPtrs, hostPtrs.data(), N * sizeof(T*), hipMemcpyHostToDevice);


}
template <typename T>
Tensor<T>::Tensor(int n, int c, int h, int w, T* hostData) :N(n), C(c), H(h), W(w)
{

	Strides[0] = 1;
	Strides[1] = W;
	Strides[2] = H * W;
	Strides[3] = C * H * W;

	IsChunkPart = false;
	IsOwnData = true;

	TotalSize = sizeof(T) * N * C * H * W;

	hipMalloc(&Data, TotalSize);
	hipMemcpy(Data, hostData, TotalSize, hipMemcpyHostToDevice);

	std::vector<T*> hostPtrs(N);
	for (size_t i = 0; i < N; ++i)
		hostPtrs[i] = Data + i * Strides[3];

	hipMalloc(&BatchPtrs, N * sizeof(T*));
	hipMemcpy(BatchPtrs, hostPtrs.data(), N * sizeof(T*), hipMemcpyHostToDevice);

}

template <typename T>
Tensor<T>::Tensor(int n, int h, int w) :N(n), C(1), H(h), W(w)
{

	Strides[0] = 1;
	Strides[1] = W;
	Strides[2] = H * W;
	Strides[3] = C * H * W;

	IsChunkPart = false;
	IsOwnData = true;

	TotalSize = sizeof(T) * N * C * H * W;

	hipMalloc(&Data, TotalSize);

	std::vector<T*> hostPtrs(N);
	for (size_t i = 0; i < N; ++i)
		hostPtrs[i] = Data + i * Strides[3];

	hipMalloc(&BatchPtrs, N * sizeof(T*));
	hipMemcpy(BatchPtrs, hostPtrs.data(), N * sizeof(T*), hipMemcpyHostToDevice);
}
template <typename T>
Tensor<T>::Tensor(int n, int h, int w, T* hostData) :N(n), C(1), H(h), W(w)
{

	Strides[0] = 1;
	Strides[1] = W;
	Strides[2] = H * W;
	Strides[3] = C * H * W;

	IsChunkPart = false;
	IsOwnData = true;

	TotalSize = sizeof(T) * N * C * H * W;

	hipMalloc(&Data, TotalSize);
	hipMemcpy(Data, hostData, TotalSize, hipMemcpyHostToDevice);

	std::vector<T*> hostPtrs(N);
	for (size_t i = 0; i < N; ++i)
		hostPtrs[i] = Data + i * Strides[3];

	hipMalloc(&BatchPtrs, N * sizeof(T*));
	hipMemcpy(BatchPtrs, hostPtrs.data(), N * sizeof(T*), hipMemcpyHostToDevice);
}

template <typename T>
Tensor<T>::Tensor(int h, int w) :N(1), C(1), H(h), W(w)
{
	Strides[0] = 1;
	Strides[1] = W;
	Strides[2] = H * W;
	Strides[3] = C * H * W;

	IsChunkPart = false;
	IsOwnData = true;

	TotalSize = sizeof(T) * N * C * H * W;

	hipMalloc(&Data, TotalSize);

	std::vector<T*> hostPtrs(N);
	for (size_t i = 0; i < N; ++i)
		hostPtrs[i] = Data + i * Strides[3];

	hipMalloc(&BatchPtrs, N * sizeof(T*));
	hipMemcpy(BatchPtrs, hostPtrs.data(), N * sizeof(T*), hipMemcpyHostToDevice);
}
template <typename T>
Tensor<T>::Tensor(int h, int w, T* hostData) :N(1), C(1), H(h), W(w)
{
	Strides[0] = 1;
	Strides[1] = W;
	Strides[2] = H * W;
	Strides[3] = C * H * W;

	IsChunkPart = false;
	IsOwnData = true;

	TotalSize = sizeof(T) * N * C * H * W;

	hipMalloc(&Data, TotalSize);
	hipMemcpy(Data, hostData, TotalSize, hipMemcpyHostToDevice);

	std::vector<T*> hostPtrs(N);
	for (size_t i = 0; i < N; ++i)
		hostPtrs[i] = Data + i * Strides[3];

	hipMalloc(&BatchPtrs, N * sizeof(T*));
	hipMemcpy(BatchPtrs, hostPtrs.data(), N * sizeof(T*), hipMemcpyHostToDevice);
}

template <typename T>
Tensor<T>::Tensor(int h) :N(1), C(1), H(h), W(1)
{
	Strides[0] = 1;
	Strides[1] = W;
	Strides[2] = H * W;
	Strides[3] = C * H * W;

	IsChunkPart = false;
	IsOwnData = true;

	TotalSize = sizeof(T) * N * C * H * W;

	hipMalloc(&Data, TotalSize);

	std::vector<T*> hostPtrs(N);
	for (size_t i = 0; i < N; ++i)
		hostPtrs[i] = Data + i * Strides[3];

	hipMalloc(&BatchPtrs, N * sizeof(T*));
	hipMemcpy(BatchPtrs, hostPtrs.data(), N * sizeof(T*), hipMemcpyHostToDevice);

}
template <typename T>
Tensor<T>::Tensor(int h, T* hostData) :N(1), C(1), H(h), W(1)
{
	Strides[0] = 1;
	Strides[1] = W;
	Strides[2] = H * W;
	Strides[3] = C * H * W;

	IsChunkPart = false;
	IsOwnData = true;

	TotalSize = sizeof(T) * N * C * H * W;

	hipMalloc(&Data, TotalSize);
	hipMemcpy(Data, hostData, TotalSize, hipMemcpyHostToDevice);

	std::vector<T*> hostPtrs(N);
	for (size_t i = 0; i < N; ++i)
		hostPtrs[i] = Data + i * Strides[3];

	hipMalloc(&BatchPtrs, N * sizeof(T*));
	hipMemcpy(BatchPtrs, hostPtrs.data(), N * sizeof(T*), hipMemcpyHostToDevice);
}


template <typename T>
Tensor<T>::~Tensor()
{
	if (IsOwnData && Data != nullptr)
	{
		hipFree(Data);
		Data = nullptr;
	}

	if (BatchPtrs != nullptr)
	{
		hipFree(BatchPtrs);
		BatchPtrs = nullptr;
	}
}



template <typename T>
Tensor<T>::Tensor(const Tensor<T>& other)
	: N(other.N), C(other.C), H(other.H), W(other.W), IsChunkPart(other.IsChunkPart), IsOwnData(true), TotalSize(other.TotalSize)
{
	memcpy(Strides, other.Strides, 4 * sizeof(int));

	hipMalloc(&Data, TotalSize);
	hipMemcpy(Data, other.Data, TotalSize, hipMemcpyDeviceToDevice);

	std::vector<T*> hostPtrs(N);
	for (size_t i = 0; i < N; ++i)
		hostPtrs[i] = Data + i * Strides[3];
	hipMalloc(&BatchPtrs, N * sizeof(T*));
	hipMemcpy(BatchPtrs, hostPtrs.data(), N * sizeof(T*), hipMemcpyHostToDevice);
}
template <typename T>
Tensor<T>& Tensor<T>::operator=(const Tensor<T>& other)
{
	if (this != &other)
	{
		if (IsOwnData && Data)
			hipFree(Data);
		if (BatchPtrs)
			hipFree(BatchPtrs);

		N = other.N;
		C = other.C;
		H = other.H;
		W = other.W;
		IsChunkPart = other.IsChunkPart;
		IsOwnData = true;
		TotalSize = other.TotalSize;
		memcpy(Strides, other.Strides, 4 * sizeof(int));

		hipMalloc(&Data, TotalSize);
		hipMemcpy(Data, other.Data, TotalSize, hipMemcpyDeviceToDevice);
		std::vector<T*> hostPtrs(N);
		for (size_t i = 0; i < N; ++i)
			hostPtrs[i] = Data + i * Strides[3];
		hipMalloc(&BatchPtrs, N * sizeof(T*));
		hipMemcpy(BatchPtrs, hostPtrs.data(), N * sizeof(T*), hipMemcpyHostToDevice);
	}
	return *this;
}

template <typename T>
Tensor<T>::Tensor(Tensor<T>&& other) noexcept
	: N(other.N), C(other.C), H(other.H), W(other.W), IsChunkPart(other.IsChunkPart), IsOwnData(other.IsOwnData), Data(other.Data), BatchPtrs(other.BatchPtrs), TotalSize(other.TotalSize)
{
	memcpy(Strides, other.Strides, 4 * sizeof(int));
	other.Data = nullptr;
	other.BatchPtrs = nullptr;
	other.IsOwnData = false;
	other.TotalSize = 0;
}
template <typename T>
Tensor<T>& Tensor<T>::operator=(Tensor<T>&& other) noexcept
{
	if (this != &other)
	{
		if (IsOwnData && Data)
			hipFree(Data);
		if (BatchPtrs)
			hipFree(BatchPtrs);

		N = other.N;
		C = other.C;
		H = other.H;
		W = other.W;
		IsChunkPart = other.IsChunkPart;
		IsOwnData = other.IsOwnData;
		TotalSize = other.TotalSize;
		memcpy(Strides, other.Strides, 4 * sizeof(int));
		Data = other.Data;
		BatchPtrs = other.BatchPtrs;

		other.Data = nullptr;
		other.BatchPtrs = nullptr;
		other.IsOwnData = false;
		other.TotalSize = 0;
	}
	return *this;
}


template <typename T>
void Tensor<T>::Fill(T value)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, Device);

	int minGridSize = 0;
	int blockSize = 0;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, FillKernel<T>, 0, 0);

	int blocksPerGrid = (TotalSize + blockSize - 1) / blockSize;

	FillKernel<T> << <blocksPerGrid, blockSize >> > (Data, TotalSize, value);

	hipDeviceSynchronize();
}

template <typename T>
void Tensor<T>::FillRandomUniform()
{
	unsigned long long seed = std::chrono::system_clock::now().time_since_epoch().count();
	FillRandomUniform(seed);

}

template <typename T>
void Tensor<T>::FillRandomUniform(unsigned long long seed)
{
	if (TotalSize == 0) return;

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, Device);

	int minGridSize = 0;
	int blockSize = 0;

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, FillRandomUniformKernel<T>, 0, 0);

	size_t numElements = TotalSize / sizeof(T);
	if (numElements == 0) return;

	int blocksPerGrid = (numElements + blockSize - 1) / blockSize;
	if (blocksPerGrid == 0 && numElements > 0) blocksPerGrid = 1;

	FillRandomUniformKernel<T> << <blocksPerGrid, blockSize >> > (Data, numElements, seed);

	hipGetLastError();

	hipDeviceSynchronize();
}



template class Tensor<float>;
template class Tensor<double>;

template class Tensor<__half>;
template class Tensor<__hip_fp8_e5m2_fnuz>;
template class Tensor<__hip_fp8_e4m3_fnuz>;
template class Tensor<__nv_fp8_e8m0>;
template class Tensor<__hip_fp4_e2m1>;