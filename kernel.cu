﻿#include <iostream>
#include "Tensor.cuh"
#include <vector>
#include <hipblas.h>

#pragma comment(lib, "cublas.lib")
#pragma comment(lib, "cudnn.lib")

template <typename T>
void Matmul(hipblasHandle_t handle, Tensor<T>& A, Tensor<T>& B, Tensor<T>& C, bool aTrans, bool bTrans, T alpha = 1, T beta = 0)
{
	int m, n, k, lda, ldb, ldc;
	hipblasStatus_t status;

	switch (A.GetDimsize()) {
	case(2):

		m = bTrans ? B.GetLen(0) : B.GetLen(1);
		n = C.GetLen(0);
		k = aTrans ? A.GetLen(0) : A.GetLen(1);

		lda = B.GetStride(1);
		ldb = A.GetStride(1);
		ldc = bTrans ? B.GetLen(0) : B.GetLen(1);

		status = hipblasGemmEx(
			handle,
			bTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			aTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			m,
			n,
			k,
			&alpha,
			B.GetData(), HIP_R_8I, lda,
			A.GetData(), HIP_R_8I, ldb,
			&beta,
			C.GetData(), C.CudaDataType, ldc,
			HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
		break;

	case(3):

		m = bTrans ? B.GetLen(1) : B.GetLen(2);
		n = C.GetLen(1);
		k = aTrans ? A.GetLen(1) : A.GetLen(2);

		lda = B.GetStride(1);
		ldb = A.GetStride(1);
		ldc = bTrans ? B.GetLen(1) : B.GetLen(2);

		hipblasGemmBatchedEx(
			handle,
			bTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			aTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			m,
			n,
			k,
			&alpha,
			B.GetBatchPtrs(), B.CudaDataType, lda,
			A.GetBatchPtrs(), A.CudaDataType, ldb,
			&beta,
			C.GetBatchPtrs(), C.CudaDataType, ldc,
			A.GetLen(0), HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

		break;

	case(4):

		m = bTrans ? B.GetLen(2) : B.GetLen(3);
		n = C.GetLen(2);
		k = aTrans ? A.GetLen(2) : A.GetLen(3);

		lda = B.GetStride(1);
		ldb = A.GetStride(1);
		ldc = bTrans ? B.GetLen(2) : B.GetLen(3);

		hipblasGemmBatchedEx(
			handle,
			bTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			aTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			m,
			n,
			k,
			&alpha,
			B.GetBatchPtrs(), B.CudaDataType, lda,
			A.GetBatchPtrs(), A.CudaDataType, ldb,
			&beta,
			C.GetBatchPtrs(), C.CudaDataType, ldc,
			A.GetLen(0) + A.GetLen(1), HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

		break;
	default:
		break;
	}

}


int main() {
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	Tensor<float> A(5, 4);
	Tensor<float> B(4, 3);
	Tensor<float> C(5, 3);

	float alpha = static_cast<float>(1.0f);
	float beta = static_cast<float>(0.0f);

	A.FillRandomUniform();
	B.FillRandomUniform();

	auto A_chunks = A.Chunk(1, 2);
	auto B_chunks = B.Chunk(1, 2);

	Matmul(handle, A, B, C, false, false, alpha, beta);

	std::cout << "A:" << A.ToString() << std::endl;
	std::cout << "B:" << B.ToString() << std::endl;
	std::cout << "C2:" << C.ToString() << std::endl;

	hipblasDestroy(handle);


	return 0;
}