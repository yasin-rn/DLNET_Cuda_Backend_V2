#include "hip/hip_runtime.h"
﻿#include <iostream>
#include "Tensor.cuh"
#include <vector>
#include <hipblas.h>

#pragma comment(lib, "cublas.lib")
#pragma comment(lib, "cudnn.lib")

template <typename T>
void Matmul(hipblasHandle_t handle, Tensor<T>& A, Tensor<T>& B, Tensor<T>& C, bool aTrans, bool bTrans, T alpha = 1, T beta = 0)
{
	int m, n, k, lda, ldb, ldc;
	hipblasStatus_t status;

	switch (A.GetDimsize()) {
	case(2):

		m = bTrans ? B.GetLen(0) : B.GetLen(1);
		n = C.GetLen(0);
		k = aTrans ? A.GetLen(0) : A.GetLen(1);

		lda = B.GetStride(1);
		ldb = A.GetStride(1);
		ldc = bTrans ? B.GetLen(0) : B.GetLen(1);

		status = hipblasGemmEx(
			handle,
			bTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			aTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			m,
			n,
			k,
			&alpha,
			B.GetData(), B.CudaDataType, lda,
			A.GetData(), A.CudaDataType, ldb,
			&beta,
			C.GetData(), C.CudaDataType, ldc,
			C.CudaDataType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
		break;

	case(3):

		m = bTrans ? B.GetLen(1) : B.GetLen(2);
		n = C.GetLen(1);
		k = aTrans ? A.GetLen(1) : A.GetLen(2);

		lda = B.GetStride(1);
		ldb = A.GetStride(1);
		ldc = bTrans ? B.GetLen(1) : B.GetLen(2);

		hipblasGemmBatchedEx(
			handle,
			bTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			aTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			m,
			n,
			k,
			&alpha,
			B.GetBatchPtrs(), B.CudaDataType, lda,
			A.GetBatchPtrs(), A.CudaDataType, ldb,
			&beta,
			C.GetBatchPtrs(), C.CudaDataType, ldc,
			A.GetLen(0), C.CudaDataType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

		break;

	case(4):

		m = bTrans ? B.GetLen(2) : B.GetLen(3);
		n = C.GetLen(2);
		k = aTrans ? A.GetLen(2) : A.GetLen(3);

		lda = B.GetStride(1);
		ldb = A.GetStride(1);
		ldc = bTrans ? B.GetLen(2) : B.GetLen(3);

		hipblasGemmBatchedEx(
			handle,
			bTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			aTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
			m,
			n,
			k,
			&alpha,
			B.GetBatchPtrs(), B.CudaDataType, lda,
			A.GetBatchPtrs(), A.CudaDataType, ldb,
			&beta,
			C.GetBatchPtrs(), C.CudaDataType, ldc,
			A.GetLen(0) + A.GetLen(1), C.CudaDataType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
		 
		break;
	default:
		break;
	}

}


int main() {
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	Tensor<__hip_fp8_e5m2_fnuz> A(5, 8);
	Tensor<__hip_fp8_e5m2_fnuz> B(4, 6);
	Tensor<__hip_fp8_e5m2_fnuz> C1(5, 3);
	Tensor<__hip_fp8_e5m2_fnuz> C2(5, 3);

	__hip_fp8_e5m2_fnuz alpha = static_cast<__hip_fp8_e5m2_fnuz>(1.0f);
	__hip_fp8_e5m2_fnuz beta = static_cast<__hip_fp8_e5m2_fnuz>(0.0f);

	A.FillRandomUniform();
	B.FillRandomUniform();

	auto A_chunks = A.Chunk(1, 2);
	auto B_chunks = B.Chunk(1, 2);

	Matmul(handle, A_chunks[0], B_chunks[0], C1, false, false, alpha, beta);
	Matmul(handle, A_chunks[1], B_chunks[1], C2, false, false, alpha, beta);

	std::cout << "A[0]:" << A_chunks[0].ToString() << std::endl;
	std::cout << "B[0]:" << B_chunks[0].ToString() << std::endl;
	std::cout << "C1:" << C1.ToString() << std::endl;

	std::cout << "A[1]:" << A_chunks[1].ToString() << std::endl;
	std::cout << "B[1]:" << B_chunks[1].ToString() << std::endl;
	std::cout << "C2:" << C2.ToString() << std::endl;

	hipblasDestroy(handle);


	return 0;
}