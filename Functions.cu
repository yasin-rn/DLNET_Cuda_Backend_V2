#include "Functions.cuh"

hipblasHandle_t Functions::CreateCublasHandle()
{
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	return handle;
}

void Functions::DestroyCublasHandle(hipblasHandle_t handle)
{
	hipblasDestroy(handle);
}

hipdnnHandle_t Functions::CreateCudnnHandle()
{
	hipdnnHandle_t handle;
	hipdnnCreate(&handle);
	return handle;
}

void Functions::DestroyCudnnHandle(hipdnnHandle_t handle)
{
	hipdnnDestroy(handle);
}

